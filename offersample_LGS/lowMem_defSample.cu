#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>


#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>



#define BLOCK_H 4
#define BLOCK_W 8
#define BLOCK_HW BLOCK_H * BLOCK_W
#define CHANNEL_STRIDE 32


__forceinline__ __device__
bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__global__ void lowMem_defSample_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> offset,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> corr,
    int r)
{
  const int b = blockIdx.x;
  const int h0 = blockIdx.y * blockDim.x;
  const int w0 = blockIdx.z * blockDim.y;
  const int tid = threadIdx.x * blockDim.y + threadIdx.y;

  const int H1 = fmap1.size(1);
  const int W1 = fmap1.size(2);
  const int H2 = fmap2.size(1);
  const int W2 = fmap2.size(2);
  const int N = coords.size(1);
  const int C = fmap1.size(3);

  __shared__ scalar_t f1[CHANNEL_STRIDE][BLOCK_HW];
  __shared__ scalar_t f2_Q11[CHANNEL_STRIDE][BLOCK_HW];
  __shared__ scalar_t f2_Q21[CHANNEL_STRIDE][BLOCK_HW];
  __shared__ scalar_t f2_Q12[CHANNEL_STRIDE][BLOCK_HW];
  __shared__ scalar_t f2_Q22[CHANNEL_STRIDE][BLOCK_HW];
  
  __shared__ float x2s[BLOCK_HW];
  __shared__ float y2s[BLOCK_HW];
  // __shared__ float ofs_x[BLOCK_HW];
  // __shared__ float ofs_y[BLOCK_HW];

  for (int c=0; c<C; c+=CHANNEL_STRIDE) {
    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      if (within_bounds(h1, w1, H1, W1))
        f1[c1][k1] = fmap1[b][h1][w1][c+c1];
      
      else
        f1[c1][k1] = 0.0;
    }

    __syncthreads();

    for (int n=0; n<N; n++) {
      int h1 = h0 + threadIdx.x;
      int w1 = w0 + threadIdx.y;
      int rd = 2*r + 1;    
      
      for (int iy=0; iy<rd; iy++) {
        for (int ix=0; ix<rd; ix++) {
            
          if (within_bounds(h1, w1, H1, W1)) {
            offset[b*n][h1][w1][static_cast<int>(rd/2)][static_cast<int>(rd/2)][0] = 0.0f;
            offset[b*n][h1][w1][static_cast<int>(rd/2)][static_cast<int>(rd/2)][1] = 0.0f;
            x2s[tid] = coords[b][n][h1][w1][0]+offset[b*n][h1][w1][ix][iy][0];
            y2s[tid] = coords[b][n][h1][w1][1]+offset[b*n][h1][w1][ix][iy][1];
          
        }
        __syncthreads();
            float dx= x2s[tid] - floor(x2s[tid]);
            float dy= y2s[tid] - floor(y2s[tid]);

          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            
            int k1 = k + tid / CHANNEL_STRIDE;
           
            int h2 = static_cast<int>(floor(y2s[k1])) - r + iy;
            int h2_high = h2+1;
            int w2 = static_cast<int>(floor(x2s[k1])) - r + ix;
            int w2_high = w2+1;
            int c2 = tid % CHANNEL_STRIDE;

            if (within_bounds(h2, w2, H2, W2))
              f2_Q11[c2][k1] = fmap2[b][h2][w2][c+c2];
            
            else
                f2_Q11[c2][k1] = static_cast<scalar_t>(0.0);
            
            if (within_bounds(h2, w2_high, H2, W2))
              f2_Q21[c2][k1] = fmap2[b][h2][w2_high][c+c2];
            
            else
                f2_Q21[c2][k1] = static_cast<scalar_t>(0.0);

            if (within_bounds(h2_high, w2, H2, W2))
              f2_Q12[c2][k1] = fmap2[b][h2_high][w2][c+c2];
            
            else
              f2_Q12[c2][k1] = static_cast<scalar_t>(0.0);

            if (within_bounds(h2_high, w2_high, H2, W2))
              f2_Q22[c2][k1] = fmap2[b][h2_high][w2_high][c+c2];
            
            else
                f2_Q22[c2][k1] = static_cast<scalar_t>(0.0);
          }

          __syncthreads();
      
          scalar_t Q11 = 0.0;
          scalar_t Q21 = 0.0;
          scalar_t Q12 = 0.0;
          scalar_t Q22 = 0.0;
          for (int k=0; k<CHANNEL_STRIDE; k++)
            {
                Q11 += f1[k][tid] * f2_Q11[k][tid];
                Q21 += f1[k][tid] * f2_Q21[k][tid];
                Q12 += f1[k][tid] * f2_Q12[k][tid];
                Q22 += f1[k][tid] * f2_Q22[k][tid];

            }
            // int ix_se = H1*W1*(iy + rd*ix);

            // scalar_t* corr_ptr = &corr[b][n][0][0][h1][w1];

          if (iy < rd && ix < rd && within_bounds(h1, w1, H1, W1))
          {
            corr[b][n][ix][iy][h1][w1] = Q11 * scalar_t((1.0f - dy) * (1.0f - dx)) + 
          Q21 * scalar_t((1.0f - dy) * dx) + 
          Q12 * scalar_t(dy * (1.0f - dx)) + 
          Q22 * scalar_t(dy * dx);
        }

        }
      } 
    }
  }
}



std::vector<torch::Tensor> lowMem_defSample_cuda(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor offset,
  int radius)
{
  const auto B = coords.size(0);
  const auto N = coords.size(1);
  const auto H = coords.size(2);
  const auto W = coords.size(3);

  const auto rd = 2 * radius + 1;
  auto opts = fmap1.options();
  auto corr = torch::zeros({B, N, rd,rd, H, W}, opts);
  
  const dim3 blocks(B, (H+BLOCK_H-1)/BLOCK_H, (W+BLOCK_W-1)/BLOCK_W);
  const dim3 threads(BLOCK_H, BLOCK_W);


  AT_DISPATCH_FLOATING_TYPES_AND_HALF(fmap1.type(), "altcorr_forward_kernel", ([&] {
    lowMem_defSample_kernel<scalar_t><<<blocks, threads>>>(
        fmap1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        fmap2.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
        offset.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        corr.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        radius);
  }));

  return {corr};
}